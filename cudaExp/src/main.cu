#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <math_functions.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <sys/time.h>
#include <omp.h>

#include "../headers/serialKnn.h"
#include "../headers/pointsHelper.h"
void printPoints(int* pts, int num , int dim);

void printTime(char* text, struct timeval end , struct timeval start);


__global__ void devKnnShared(float* points, float* queries, int* points_per_block, int* queries_per_block, int* res_indexes , float* res_dists, int number_of_queries , int max_points)
{
    int b = blockIdx.x*gridDim.y*gridDim.z + blockIdx.y*gridDim.z + blockIdx.z;
    int num_of_queries = queries_per_block[b];
    int mulq = 0;
    int integral_queries = 0;
    int integral_points = 0;
    int qrs_shifter;
    int num_of_points ;
    float nbrs_dist;
    int nbrs_indx;
    int mulp ;
    int grid_d = gridDim.x;
    __shared__ float sh_pts[532][3];
    int flag = 0;
    float myQuery[3];
    for(int i = 0; i < b; i++){
        integral_points += points_per_block[i];
        integral_queries += queries_per_block[i];
    }
    while(mulq*blockDim.x < num_of_queries){
        int q = mulq*blockDim.x + threadIdx.x;
        qrs_shifter = integral_queries + q;
        num_of_points = points_per_block[b];
        nbrs_dist = 100;
        nbrs_indx = 1;
        mulp = 0;
        if(q < num_of_queries){
            myQuery[0] = queries[qrs_shifter*3 + 0];
            myQuery[1] = queries[qrs_shifter*3 + 1];
            myQuery[2] = queries[qrs_shifter*3 + 2];
        }
        while(mulp*blockDim.x < num_of_points){
            int p2 = mulp*blockDim.x + threadIdx.x;
            int pts_shifter2 = integral_points + p2;

            __syncthreads();
            if(p2 < num_of_points && pts_shifter2 < max_points){
                for(int d = 0; d < 3; d++){
                    sh_pts[threadIdx.x][d] = points[pts_shifter2*3 + d];
                }
            }
            __syncthreads();

            if(q < num_of_queries){
                int limit = min(num_of_points,(mulp+1)*blockDim.x);
                for(int p = mulp*blockDim.x; p < limit; p++){
                    int pts_shifter = integral_points + p;
                    float dist = 0;
                    for(int d = 0; d < 3; d++){
                        dist += powf(myQuery[d]- sh_pts[p - mulp*blockDim.x][d] ,2);   //points[pts_shifter*3+d],2); //   
                    }
                    dist = sqrtf(dist);
                    if(dist < nbrs_dist){
                        nbrs_dist = dist;
                        nbrs_indx = pts_shifter;
                    }  
                }
            }
            mulp++;
        }

        if(q < num_of_queries){
            if(nbrs_dist < res_dists[qrs_shifter]){
                res_dists[qrs_shifter] = nbrs_dist;
                res_indexes[qrs_shifter] = nbrs_indx;
            }
        }
               
        
       

        mulq++;
    }

    // Search neighbour blocks

    int nbrs_blocks[27];
    int number_of_nbrs_blocks = 0; 
    for(int i = -1; i <= 1; i++){
        for(int j = -1; j <= 1; j++){
            for(int k = -1; k <= 1; k++){
                if(i != 0 | j != 0 | k != 0){
                    int nx = blockIdx.x + i;
                    int ny = blockIdx.y  + j;
                    int nz = blockIdx.z  + k;

                    if(!(nx<0 | ny<0 | nz<0 | nx >=grid_d | ny >= grid_d | nz>=grid_d)){
                        nbrs_blocks[number_of_nbrs_blocks] = nx*grid_d*grid_d + ny*grid_d + nz;
                        number_of_nbrs_blocks++;
                    }
                }
            }
        }
    }
    for(int nb = 0; nb < number_of_nbrs_blocks; nb++){
        integral_points = 0;
        for(int i = 0; i < nbrs_blocks[nb]; i++){
            integral_points += points_per_block[i];
        }
        mulq = 0;
        while(mulq*blockDim.x < num_of_queries){
            int q = mulq*blockDim.x + threadIdx.x;
            qrs_shifter = integral_queries + q;
            num_of_points = points_per_block[nbrs_blocks[nb]];
            nbrs_dist = 100;
            nbrs_indx = 1;
            mulp = 0;
            if(q < num_of_queries){
                myQuery[0] = queries[qrs_shifter*3 + 0];
                myQuery[1] = queries[qrs_shifter*3 + 1];
                myQuery[2] = queries[qrs_shifter*3 + 2];
            }
            
            while(mulp*blockDim.x < num_of_points){
                int p2 = mulp*blockDim.x + threadIdx.x;
                int pts_shifter2 = integral_points + p2;

                __syncthreads();
                if(p2 < num_of_points && pts_shifter2 < max_points){
                    for(int d = 0; d < 3; d++){
                        sh_pts[threadIdx.x][d] = points[pts_shifter2*3 + d];
                    }
                }
                __syncthreads();

                if(q < num_of_queries){
                    int limit = min(num_of_points,(mulp+1)*blockDim.x);
                    for(int p = mulp*blockDim.x; p < limit; p++){
                        int pts_shifter = integral_points + p;
                        float dist = 0;
                        for(int d = 0; d < 3; d++){
                            dist += powf(myQuery[d]- sh_pts[p - mulp*blockDim.x][d] ,2);   //points[pts_shifter*3+d],2); //   
                        }
                        dist = sqrtf(dist);
                        if(dist < nbrs_dist){
                            nbrs_dist = dist;
                            nbrs_indx = pts_shifter;
                        }  
                    }
                }
                mulp++;
            }

            if(q < num_of_queries){
                if(nbrs_dist < res_dists[qrs_shifter]){
                    res_dists[qrs_shifter] = nbrs_dist;
                    res_indexes[qrs_shifter] = nbrs_indx;
                }
            }
                
            
        

            mulq++;
        }
    }
}

/******************** INPUT *********************
 * 1st param -> number of points    (default 2^5)
 * 2nd param -> grid dimensions     (default 2^1)
 * 3rd param -> seed                (default 1,2)
*************************************************/
int main(int argc, char** argv){

    hipDeviceReset();
    struct timeval totalProgramStart,totalProgramEnd,tstart,tend;
    gettimeofday(&totalProgramStart,NULL);
    int *cudaInit;
    //----------------------------------------------//
    
    int number_of_points = 5;
    int grid_d = 1;
    int k_num = 1;
    int seed = 1;
    if(argc > 1){
        number_of_points = atoi(argv[1]);
    }
    number_of_points = pow(2,number_of_points);
    if(argc > 2){
        grid_d = atoi(argv[2]);
    }
    grid_d = pow(2,grid_d);
    if(argc > 3){
        k_num = atoi(argv[3]);
    }
    if(argc > 4){
        seed = atoi(argv[4]);
    }
    int number_of_queries = number_of_points;
    int dimensions = 3;
    float side_block_length = ((float)1)/((float)grid_d);
    printf("Number of points:%d\nNumber of queries:%d\nDimensions:%d\nGrid Dimensions:%d\nK for k-nn:%d\nSideBlock Length%f\n",
                                                number_of_points,number_of_queries,dimensions,grid_d,k_num,side_block_length);
    gettimeofday(&tstart,NULL);
    float* points = (float*)malloc(number_of_points*dimensions*sizeof(float));
    float* queries = (float*)malloc(number_of_queries*dimensions*sizeof(float));
    float* grid_arranged_points = (float*)malloc(number_of_points*dimensions*sizeof(float));
    float* grid_arranged_queries = (float*)malloc(number_of_queries*dimensions*sizeof(float));
    int* block_of_point = (int*)malloc(number_of_points*dimensions*sizeof(int));
    int* block_of_query = (int*)malloc(number_of_queries*dimensions*sizeof(int));
    int* points_per_block = (int*)malloc(grid_d*grid_d*grid_d*sizeof(int));
    int* queries_per_block = (int*)malloc(grid_d*grid_d*grid_d*sizeof(int));
    int* integral_points_per_block = (int*)malloc(grid_d*grid_d*grid_d*sizeof(int));
    int* integral_queries_per_block = (int*)malloc(grid_d*grid_d*grid_d*sizeof(int));

    
    float* knns = (float*) malloc(number_of_queries*dimensions*sizeof(int));
    float* knns_gpu = (float*) malloc(number_of_queries*dimensions*sizeof(int));
    float* knns_dists = (float*)malloc(number_of_queries*sizeof(float));

    for(int i = 0; i < number_of_queries; i++){
        knns_dists[i] = 100;
    }

    gettimeofday(&tend,NULL);
    printTime("CPU MALLOC TIME ",tend,tstart);

    gettimeofday(&tstart,NULL);
    generatePoints(points, number_of_points, dimensions, 0, 1, 1);
    generatePoints(queries, number_of_queries, dimensions, 0, 1, 2);
    gettimeofday(&tend,NULL);
    printTime("GENERATION TIME ",tend,tstart);

    gettimeofday(&tstart,NULL);
    assignPointsToBlocks(points, block_of_point , points_per_block , side_block_length , number_of_points, grid_d , dimensions);
    assignPointsToBlocks(queries, block_of_query , queries_per_block , side_block_length , number_of_queries, grid_d , dimensions);

    for(int i = 0; i < grid_d*grid_d*grid_d; i++){
        integral_points_per_block[i] = 0;
        integral_queries_per_block[i] = 0;
        for(int j = 0; j < i; j++){
            integral_points_per_block[i] += points_per_block[j];
            integral_queries_per_block[i] += queries_per_block[j];
        }
    }

    
    rearrangePointsToGrid(points,grid_arranged_points, block_of_point , points_per_block , side_block_length , number_of_points, grid_d , dimensions);
    rearrangePointsToGrid(queries,grid_arranged_queries, block_of_query , queries_per_block , side_block_length , number_of_queries, grid_d , dimensions);

    assignPointsToBlocks(grid_arranged_points, block_of_point , points_per_block , side_block_length , number_of_points, grid_d , dimensions);
    assignPointsToBlocks(grid_arranged_queries, block_of_query , queries_per_block , side_block_length , number_of_queries, grid_d , dimensions);

    gettimeofday(&tend,NULL);
    printTime("CPU BINNING TIME ",tend,tstart);

    
  gettimeofday(&tstart,NULL);
    float* dev_points;
    hipError_t cuer;
    cuer = hipMalloc(&dev_points,number_of_points*3*sizeof(float));
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(dev_points, grid_arranged_points, number_of_points*3*sizeof(float),hipMemcpyHostToDevice);
    printf("%s\n",hipGetErrorName(cuer));
    float* dev_queries;
    cuer = hipMalloc(&dev_queries, number_of_queries*3*sizeof(float));
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(dev_queries,grid_arranged_queries,number_of_queries*3*sizeof(float),hipMemcpyHostToDevice);
    printf("%s\n",hipGetErrorName(cuer));
    int* dev_points_per_block;
    cuer = hipMalloc(&dev_points_per_block, grid_d*grid_d*grid_d*sizeof(int));
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(dev_points_per_block , points_per_block, grid_d*grid_d*grid_d*sizeof(int),hipMemcpyHostToDevice);
    printf("%s\n",hipGetErrorName(cuer));
    int* dev_queries_per_blcok;
    cuer = hipMalloc(&dev_queries_per_blcok, grid_d*grid_d*grid_d*sizeof(int));
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(dev_queries_per_blcok, queries_per_block ,grid_d*grid_d*grid_d*sizeof(int),hipMemcpyHostToDevice);
    printf("%s\n",hipGetErrorName(cuer));



    float* res_dists = (float*)malloc(number_of_queries*sizeof(float));
    int* res_indexes = (int*)malloc(number_of_queries*sizeof(int));

    float* res_dists2 = (float*)malloc(number_of_queries*sizeof(float));
    int* res_indexes2 = (int*)malloc(number_of_queries*sizeof(int));
    for(int i = 0; i < number_of_queries; i++){
        res_dists[i] = 100;
        res_indexes[i] = 19;
    }
    int* dev_res_indexes;
    cuer = hipMalloc(&dev_res_indexes,number_of_queries*sizeof(int));
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(dev_res_indexes,res_indexes, number_of_queries*sizeof(int),hipMemcpyHostToDevice);
    printf("%s\n",hipGetErrorName(cuer));
    
    float *dev_res_dists;
    cuer = hipMalloc(&dev_res_dists,number_of_queries*sizeof(float));
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(dev_res_dists,res_dists,number_of_queries*sizeof(float) , hipMemcpyHostToDevice);
    printf("%s\n",hipGetErrorName(cuer));

    gettimeofday(&tend,NULL);
    printTime("GPU MALLOC",tend,tstart);

    

    //dbgKnn<<<1000,500>>>(dev_res_dists,dev_res_indexes,number_of_queries);
    devKnnShared<<<dim3(grid_d,grid_d,grid_d),512>>>(dev_points,dev_queries,dev_points_per_block , dev_queries_per_blcok,dev_res_indexes , dev_res_dists , number_of_queries,number_of_points);
    cuer = hipGetLastError();
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(res_dists,dev_res_dists,number_of_queries*sizeof(float) , hipMemcpyDeviceToHost);
    printf("%s\n",hipGetErrorName(cuer));
    cuer = hipMemcpy(res_indexes,dev_res_indexes, number_of_queries*sizeof(int),hipMemcpyDeviceToHost);
    printf("%s\n",hipGetErrorName(cuer));


         for(int i = 0; i < number_of_queries; i++){
        memcpy(&knns_gpu[i*3], &grid_arranged_points[res_indexes[i]*3], 3*sizeof(float));
    }
    printPointsToCsv("knn2.csv" , "w" , knns_gpu , number_of_queries , dimensions);
    gettimeofday(&tstart,NULL);
    printTime("GPU KNN ",tstart,tend);
   


    printPointsToCsv("knn.csv" , "w" , knns_gpu , number_of_queries*k_num , dimensions);
    printPointsToCsv("points.csv" , "w" , points , number_of_points , dimensions);
    printPointsToCsv("queries.csv" , "w" , queries , number_of_queries , dimensions);
    printPointsToCsv("points_arranged.csv" ,"w" , grid_arranged_points , number_of_points , dimensions);
    printPointsToCsv("queries_arranged.csv" , "w" , grid_arranged_queries , number_of_queries , dimensions);

    






    //debugGPUKnnGlobal(grid_arranged_points,0,0,grid_arranged_queries,0,0,points_per_block,queries_per_block,grid_d,num_of_threads,1,indxs,dsts);














    free(points);
    free(queries);
    free(grid_arranged_points);
    free(grid_arranged_queries);
    free(block_of_point);
    free(block_of_query);
    free(points_per_block);
    free(queries_per_block);
    //free(knn_res);
    //----------------------------------------------------------------//
    gettimeofday(&totalProgramEnd,NULL);
    printTime("total program time ", totalProgramEnd,totalProgramStart);
    hipProfilerStop();
    hipDeviceReset();
    return 0;
}






void printPoints(int* pts, int num, int dim){
    for(int i = 0; i < num; i++){
        printf("Points%d:\t",i);
        for(int j = 0; j < dim; j++){
            printf("x%d %d\t",j,pts[i*dim + j]);
        }
        printf("\n");
    }
}


void printTime(char* text, struct timeval end , struct timeval start){
    printf("%s ",text);
    long s=end.tv_sec-start.tv_sec;
    long us=end.tv_usec - start.tv_usec;
    if(us < 0){
        us = 1000000+us;
        s = s-1;
    }
    printf("%ld s, %ld us\n",s,us);
}
